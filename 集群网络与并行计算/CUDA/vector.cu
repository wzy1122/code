#include<stdio.h>
#include<hip/hip_runtime.h>
 
__global__ void VecAdd(float *A, float *B, float *C)
{
     int i = threadIdx.x;
      C[i] = A[i] + B[i];
}
 
int main(void)
{
    int host_A[30], host_B[30], host_C[30];      //申请主机内存，并进行初始化
    
    for(int i=0; i<30;i++)
    {
        host_A[i] = i;
        host_B[i]=i+i-1;
    }

     hipError_t err = hipSuccess;        //定义cudaError,默认为cudaSuccess(0)
 
    float *A, *B, *C;
    err = hipMalloc( (void**)&A, sizeof(float)*30);    //在GPU上申请内存
    err = hipMalloc( (void**)&B, sizeof(float)*30);
    err = hipMalloc( (void**)&C, sizeof(float)*30);
 
   hipMemcpy(A, host_A, sizeof(host_A), hipMemcpyHostToDevice);
   hipMemcpy(B, host_B, sizeof(host_B), hipMemcpyHostToDevice);
 
   VecAdd<<<1,30>>>(A, B, C);
  
   hipMemcpy(&host_C, C, sizeof(host_C), hipMemcpyDeviceToHost);
 
   for(int i=0; i<30; i++)
      printf("A[%d] + B[%d] = %d + %d = %d\n", i, i, host_A[i], host_B[i], host_C[i]);
 
   hipFree(A);
   hipFree(B);
   hipFree(C);
  
  return 0;
}
