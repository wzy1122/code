#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

const int N = 6;
__global__ void MatMulAdd(float* A, float* B, float* C, float* D, int width) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	D[y * width + x] = 0.0f;
	for (int k = 0; k < width; k++) {
		D[y * width + x] += A[y * width + k] + B[k * width + x];
	}
	D[y * width + x] += C[y * width + x];
}

__global__ void MatVecDistance(float* vec, float* mat, float* res, int N) {
	int x = threadIdx.x;
	float total = 0.0f;
	for (int k = 0; k < N; k++) {
		total += vec[k] * mat[x * N + k];
	}
	res[x] = sqrt(total);
}

void func2() {
	const int N = 6;
	float vec[N] = { 0 };
	float mat[N*N] = { 0 };
	float res[N] = { 0 };

	for (int i = 0; i < N; i++) {
		vec[i] = i;
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			mat[i * N + j] = i * N + j;
		}
	}
	
	int vecSize = sizeof(float) * N;
	int matSize = sizeof(float) * N * N;
	float* d_vec;
	float* d_mat;
	float* d_res;
	
	hipMalloc(&d_vec, vecSize);
	hipMalloc(&d_mat, matSize);
	hipMalloc(&d_res, vecSize);

	hipMemcpy(d_vec, vec, vecSize, hipMemcpyHostToDevice);
	hipMemcpy(d_mat, mat, matSize, hipMemcpyHostToDevice);
	hipMemcpy(d_res, res, vecSize, hipMemcpyHostToDevice);

	MatVecDistance << <1, N >> > (d_vec, d_mat, d_res, N);

	hipMemcpy(res, d_res, vecSize, hipMemcpyDeviceToHost);

	hipFree(d_vec);
	hipFree(d_mat);
	hipFree(d_res);

	for (int i = 0; i < N; i++) {
		printf("%f ", res[i]);
	}
} 

int main() {
	func2();
}
