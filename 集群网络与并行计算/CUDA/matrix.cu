#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

const int N = 6;
__global__ void MatMulAdd(float* A, float* B, float* C, float* D, int width) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	D[y * width + x] = 0.0f;
	for (int k = 0; k < width; k++) {
		D[y * width + x] += A[y * width + k] + B[k * width + x];
	}
	D[y * width + x] += C[y * width + x];
}

void func1() {

	float A[N * N] = { 0 };
	float B[N * N] = { 0 };
	float C[N * N] = { 0 };
	float D[N * N] = { 0 };
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			// scanf("%f", &A[i][j]);
			A[i * N + j] = i+j;
		}
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			// scanf("%f", &B[i][j]);
			B[i * N + j] =2*i+j;
		}
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			// scanf("%f", &B[i][j]);
			C[i * N + j] = i+3*j;
		}
	}

	float* d_A;
	float* d_B;
	float* d_C;
	float* d_D;

	hipMalloc(&d_A, sizeof(float) * N * N);
	hipMalloc(&d_B, sizeof(float) * N * N);
	hipMalloc(&d_C, sizeof(float) * N * N);
	hipMalloc(&d_D, sizeof(float) * N * N);

	hipMemcpy(d_A, A, sizeof(float) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(float) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(d_D, D, sizeof(float) * N * N, hipMemcpyHostToDevice);

	int num_blocks = 1;
	dim3 threads_per_block(N, N);
	MatMulAdd << <num_blocks, threads_per_block >> > (d_A, d_B, d_C, d_D, N);

	hipMemcpy(D, d_D, sizeof(float) * N * N, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%f ", D[i*N + j]);
		}
		printf("\n");
	}
}

__global__ void MatVecDistance(float* vec, float* mat, float* res, int N) {
	int x = threadIdx.x;
	float total = 0.0f;
	for (int k = 0; k < N; k++) {
		total += vec[k] * mat[x * N + k];
	}
	res[x] = sqrt(total);
}

void func2() {
	const int N = 6;
	float vec[N] = { 0 };
	float mat[N*N] = { 0 };
	float res[N] = { 0 };

	for (int i = 0; i < N; i++) {
		vec[i] = i;
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			mat[i * N + j] = i * N + j;
		}
	}
	
	int vecSize = sizeof(float) * N;
	int matSize = sizeof(float) * N * N;
	float* d_vec;
	float* d_mat;
	float* d_res;
	
	hipMalloc(&d_vec, vecSize);
	hipMalloc(&d_mat, matSize);
	hipMalloc(&d_res, vecSize);

	hipMemcpy(d_vec, vec, vecSize, hipMemcpyHostToDevice);
	hipMemcpy(d_mat, mat, matSize, hipMemcpyHostToDevice);
	hipMemcpy(d_res, res, vecSize, hipMemcpyHostToDevice);

	MatVecDistance << <1, N >> > (d_vec, d_mat, d_res, N);

	hipMemcpy(res, d_res, vecSize, hipMemcpyDeviceToHost);

	hipFree(d_vec);
	hipFree(d_mat);
	hipFree(d_res);

	for (int i = 0; i < N; i++) {
		printf("%f ", res[i]);
	}
} 

int main() {
	func1();
}
