
#include <hip/hip_runtime.h>
#include<iostream>                                                                                                       
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
 
using namespace std;
 
const int N = 10;
const int D = 8;
const int MAX = 10;
 
__global__ void cal_dis(int *train_data, int *test_data, int *dis,int pitch)
{
	int tid = blockIdx.x;
	if(tid<N)
	{
		int temp = 0;
		int sum = 0;
		for(int i=0;i<D;i++)
		{
			temp = *((int*)((char*)train_data + tid * pitch) + i) - test_data[i];
			sum += temp * temp;
		}
		dis[tid] = sum;
	}
}
 
void print(int data[][D])
{
	cout<<"training data:"<<endl;
 	for(int i=0;i<N;i++)
	{
		for(int j=0;j<D;j++)
		{
			cout<<*(*(data+i)+j)<<" ";		
		}
		cout<<endl;
	}
}
 
void print(int *data,int n)
{
	for(int i=0;i<n;i++)
	{
		cout<<data[i]<<" ";
	}
	cout<<endl;
}
 
int main()
{
	int h_train_data[N][D] , h_test_data[D] , distance[N];
 
	int *d_train_data , *d_test_data , *d_dis;
 
	size_t pitch_d;
	size_t pitch_h = D * sizeof(int) ;
 
	//allocate memory on GPU 
	hipMallocPitch( &d_train_data , &pitch_d , D * sizeof(float) , N ); 
	hipMalloc( (void**)&d_test_data ,  D*sizeof(int) );
	hipMalloc( (void**)&d_dis , 		  N*sizeof(int) );
 
	//initialize training data
	srand( (unsigned)time(NULL) );
	for( int i=0;i<N;i++ )
	{
		for( int j=0;j<D;j++)
		{
			h_train_data[i][j] = rand()%MAX;
		}
	}
	print(h_train_data);
 
	//initialize testing data
	for( int j=0;j<D;j++ )
	{
	  	h_test_data[j] = rand() % MAX;
	}
	cout<<"testing data:"<<endl;
	print(h_test_data,D);
 
	//copy training and testing data from host to device
	hipMemcpy2D( d_train_data , pitch_d , h_train_data , pitch_h , D * sizeof(int) , N , hipMemcpyHostToDevice );
	hipMemcpy( d_test_data,  h_test_data ,  D*sizeof(int), hipMemcpyHostToDevice);
 
	//calculate the distance
	cal_dis<<<N,1>>>( d_train_data,d_test_data,d_dis,pitch_d );
 
	//copy distance data from device to host
	hipMemcpy( distance , d_dis  , N*sizeof(int) , hipMemcpyDeviceToHost);
 
	cout<<"distance:"<<endl;;
	print(distance , N);
 
	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
 
	return 0;
}               