#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrixMulKernel(float *D, float *A, float *B, int width, int height){
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;
  if(tx >= width || ty >= height)
    return;

  float sum1 = 0;
  for(int i=0; i<width; ++i){
    sum1 += A[ty * width + i] * B[i * width + tx];
  }
  D[ty * width + tx] = sum1;
}

__global__ void sumMatrixOnDevice(float *E, float *D, float *C, int width, int height)
{
	int tx = threadIdx.x + blockDim.x*blockIdx.x;
	int ty = threadIdx.y + blockDim.y*blockIdx.y;
	unsigned int idx = iy * width + ix;
	//unsigned int t_n = gridDim.x*blockDim.x + gridDim.y*blockDim.y;
	if (tx < width && ty < height)
	{
		E[idx] = D[idx] + C[idx];
	
	}
}

void constantInit(float *data, int size, float val){
    for (int i = 0; i < size; ++i){
        data[i] = val;
    }
}



void matrixMul(){
  unsigned int width = 128;
  unsigned int height = 128;
  unsigned int size = width * height * sizeof(float);
  float *h_A = (float*)malloc(size);
  float *h_B = (float*)malloc(size);
  float *h_C = (float*)malloc(size);
  float *h_D = (float*)malloc(size);
  float *h_E = (float*)malloc(size);

  // Initialize host memory
  const float valB = 0.01f;
  constantInit(h_A, width*height, 1.0f);
  constantInit(h_B, width*height, valB);
  constantInit(h_c, width*height, valB);


  float *d_A, *d_B, *d_C,*d_D ,*d_E;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);
  hipMalloc((void**)&d_D, size);
  hipMalloc((void**)&d_E, size);

  //copy host memory to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

  //config dims
  dim3 block(16, 16);
  dim3 grid(width / block.x, height / block.y);

  // Excute the kernel
  matrixMulKernel<<<grid, block>>>(d_D, d_A, d_B,width, height);
  sumMatrixOnDevice<<<grid, block>>>(d_E, d_D, d_C,width, height);
  // Copy the memory from device to host
  hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_E, d_E, size, hipMemcpyDeviceToHost);

  printf("Checking computed result for correctness: ");
  bool correct = true;
  // test relative error by the formula
  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1.e-6 ; // machine zero

  for (int i = 0; i < width*height; i++){
      double abs_err = fabs(h_E[i] - (width * valB));
      double dot_length = width;
      double abs_val = fabs(h_E[i]);
      double rel_err = abs_err/abs_val/dot_length ;
      if (rel_err > eps)
      {
          printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_E[i], (float)(width*height), eps);
          correct = false;
      }
  }
  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");


  // Free
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_D);
  free(h_E);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_D);
  hipFree(d_E);
  

}

int main(){
  matrixMul();
}
